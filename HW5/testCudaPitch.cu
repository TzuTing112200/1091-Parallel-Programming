#include <stdio.h>
#include <hip/hip_runtime.h>



__global__ void test(float* devPtr, size_t pitch, int width, int height)
{
    printf("%d\t%d\t%d\t%d\n", blockIdx.x, threadIdx.x, blockIdx.y, threadIdx.y);
    float* row = (float*)((char*)devPtr + blockIdx.x * pitch); 

    row[0] = blockIdx.x + 1000000; 
}

int main() {
    int n_thread = 16;
    int width=150;
    int height=20;
    float *myMatrix = new float[width * height];

    //cudaHostAlloc((void**)&myMatrix, width * height * sizeof(float),cudaHostAllocDefault);

    memset(myMatrix, 0, width * height * sizeof(float));

    for (int i=0;i<width;i++)
    {
        for(int j=0;j<height;j++)
        {
            myMatrix[i + j * width]=j * 1000 + i;
            printf("%.0f\t",myMatrix[i + j * width]);
        }
        printf("\n");
    }
    printf("\n");

    float *dev_mat;
    size_t pitch;

    printf("%d\n",pitch);
    hipMallocPitch(&dev_mat,&pitch,width*sizeof(float),height);
    printf("%d\n%d\n\n", pitch, pitch / sizeof(float));

    hipMemcpy(dev_mat,myMatrix,width*height*sizeof(float),hipMemcpyHostToDevice);
    
    dim3 blockSize(n_thread, n_thread);
    dim3 numBlock(width / n_thread, height / n_thread);

    test<<<numBlock, blockSize>>>(dev_mat,pitch,width,height);

    hipDeviceSynchronize();

    hipMemcpy(myMatrix,dev_mat,width*height*sizeof(float),hipMemcpyDeviceToHost);

    hipFree(dev_mat);

    for (int i=0;i<width;i++)
    {
        for(int j=0;j<height;j++)
        {
            printf("%.0f\t",myMatrix[i + j * width]);
        }
        printf("\n");
    }

    return 0;
}
