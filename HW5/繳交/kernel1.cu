#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
__device__ int cal(float x, float y, int maxIterations){
    float z_re = x, z_im = y;
    int i;
    for (i = 0; i < maxIterations; ++i)
    {
        if (z_re * z_re + z_im * z_im > 4.f)
        break;

      float new_re = z_re * z_re - z_im * z_im;
      float new_im = 2.f * z_re * z_im;
      z_re = x + new_re;
      z_im = y + new_im;
    }
    return i;
}

__global__ void mandelKernel(int *img, float lowerX, float lowerY, float stepX, float stepY, int maxIterations, int resX, int resY) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    
    if(thisX < resX && thisY < resY)
    {
        float x = lowerX + thisX * stepX;
        float y = lowerY + thisY * stepY;
    
        int idx = thisY * resX + thisX;
        img[idx] = cal(x, y, maxIterations);
    }
}

// Host front-end function that allocates the memory and launches the GPU kernel 
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    int size = resX * resY * sizeof(int);
    int n_thread = 16;
    
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    
    int *h_img = (int*)malloc(size);
    
    int *d_img;
    hipMalloc((void **)&d_img, size);
    
    dim3 blockSize(n_thread, n_thread);
    dim3 numBlock(resX / n_thread, resY / n_thread);

    mandelKernel<<<numBlock, blockSize>>>(d_img, lowerX, lowerY, stepX, stepY, maxIterations, resX, resY);

    hipDeviceSynchronize();

    hipMemcpy(h_img, d_img, size, hipMemcpyDeviceToHost);
    memcpy(img, h_img, size);
    
    hipFree(d_img);
	free(h_img);
}
