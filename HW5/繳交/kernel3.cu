#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
__device__ int cal(float x, float y, int maxIterations){
    float z_re = x, z_im = y;
    int i;
    for (i = 0; i < maxIterations; ++i)
    {

        if (z_re * z_re + z_im * z_im > 4.f)
        break;

      float new_re = z_re * z_re - z_im * z_im;
      float new_im = 2.f * z_re * z_im;
      z_re = x + new_re;
      z_im = y + new_im;
    }
    return i;
}

__global__ void mandelKernel(int *img, size_t pitch, float lowerX, float lowerY, float stepX, float stepY, int maxIterations, int resX, int resY) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
	
    int thisX;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
	
	//int index = blockIdx.x * (pitch / sizeof(int)) + threadIdx.x;
    for(int i = 0; i < 16; i++)
	{
		thisX = blockIdx.x * blockDim.x + i * 256 + threadIdx.x + i;
		if(thisX < resX && thisY < resY)
		{
			float x = lowerX + thisX * stepX;
			float y = lowerY + thisY * stepY;
		
			int idx = thisY * resX + thisX;
			img[idx] = cal(x, y, maxIterations);
		}
	}
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    int size = resX * resY * sizeof(int);
    int n_thread = 16;
	
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    
    int *h_img;
	hipHostAlloc((void**)&h_img, size,hipHostMallocDefault);
	
    int *d_img;
    size_t pitch;
    hipMallocPitch(&d_img, &pitch, resX * sizeof(int), resY);
    
    dim3 blockSize(n_thread, n_thread);
    dim3 numBlock(resX / n_thread / 4, resY / n_thread);

    mandelKernel<<<numBlock, blockSize>>>(d_img, pitch, lowerX, lowerY, stepX, stepY, maxIterations, resX, resY);

    hipDeviceSynchronize();
	
    hipMemcpy(h_img, d_img, size, hipMemcpyDeviceToHost);
    memcpy(img, h_img, size);
	
    hipFree(d_img);
	hipFree(h_img);
}
