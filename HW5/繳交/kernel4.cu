#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
__device__ int cal(float x, float y, int maxIterations){
    float z_re = x, z_im = y;
    int i;
    for (i = 0; i < maxIterations; ++i)
    {
        if (z_re * z_re + z_im * z_im > 4.f)
        break;

      float new_re = z_re * z_re - z_im * z_im;
      float new_im = 2.f * z_re * z_im;
      z_re = x + new_re;
      z_im = y + new_im;
    }
    return i;
}

__global__ void mandelKernel(int *img, float lowerX, float lowerY, float stepX, float stepY, int maxIterations, int resX, int resY) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(thisX < resX * resY)
    {
        float x = lowerX + thisX % resX * stepX;
        float y = lowerY + thisX / resX * stepY;
    
        //int idx = thisY * resX + thisX;
        img[thisX] = cal(x, y, maxIterations);
    }
}

// Host front-end function that allocates the memory and launches the GPU kernel 
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    int size = resX * resY * sizeof(int);
    int n_thread = 256;
    
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    
    int *d_img;
    hipMalloc((void **)&d_img, size);
    
    // dim3 blockSize(n_thread, n_thread);
    // dim3 numBlock(resX / n_thread, resY / n_thread);

    mandelKernel<<<(resX * resY + 255) / n_thread, n_thread>>>(d_img, lowerX, lowerY, stepX, stepY, maxIterations, resX, resY);

    hipDeviceSynchronize();

    hipMemcpy(img, d_img, size, hipMemcpyDeviceToHost);
    
    hipFree(d_img);
}
