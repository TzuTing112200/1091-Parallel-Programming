#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(float *outputImage, int filterWidth, float *filter, int imageHeight, int imageWidth, float *inputImage) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int x, xe, y, l, yl, index, temp_x, temp_f;
    float4 sum, sub_sum;

    // printf("-%f, %f, %f, %f, %f, %f, %f, %f, %f, %f\n", inputImage[0], inputImage[1], inputImage[2], inputImage[3], inputImage[4], inputImage[5], inputImage[6], inputImage[7], inputImage[8], inputImage[9]);
    index = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
    // printf("-%d-%d-%d-%f-\n", blockIdx.x, threadIdx.x, index, sum);
    // outputImage[0] = 99;
    // outputImage[1] = 99;
    // outputImage[2] = 99;
    // outputImage[3] = 99;
    // outputImage[4] = 99;
    // outputImage[5] = 99;
    // outputImage[6] = 99;
    // outputImage[7] = 99;
    // outputImage[8] = 99;
    // outputImage[9] = 99;
    // printf("--%f, %f, %f, %f, %f, %f, %f, %f, %f, %f\n", outputImage[0], outputImage[1], outputImage[2], outputImage[3], outputImage[4], outputImage[5], outputImage[6], outputImage[7], outputImage[8], outputImage[9]);

    // printf("\n\n-%d-%d-%d-\n\n", blockIdx.x, blockDim.x, threadIdx.x);
    x = index / imageWidth - filterWidth;
    xe = index / imageWidth + filterWidth;
    y = index % imageWidth;
    
    sum = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    temp_f = 0;
    temp_x = x * imageWidth;

    for (; x <= xe; x++, temp_x += imageWidth)
    {
        for (l = -filterWidth; l <= filterWidth; l++, temp_f++)
        {
            if (filter[temp_f] == 0) continue;
            if (x < 4 || x + 4 >= imageHeight) continue;
            
            yl = y + l;
            if (yl < 4 || yl + 4 >= imageWidth) continue;
            
            yl += temp_x;
            sub_sum = make_float4(inputImage[yl], inputImage[yl + 1], inputImage[yl + 2], inputImage[yl + 3]);
            sub_sum *= filter[temp_f];
            
            sum += sub_sum;
        }
    }
    // printf("-%d-%d-%d-%f-\n", blockIdx.x, threadIdx.x, index, sum);
    outputImage[index++] = sum.x;
    outputImage[index++] = sum.y;
    outputImage[index++] = sum.z;
    outputImage[index] = sum.w;
}

// Host front-end function that allocates the memory and launches the GPU kernel 
void hostFE (int filterWidth, float *filter, int imageHeight, int imageWidth, float *inputImage, float *outputImage)
{
    int size = imageHeight * imageWidth * sizeof(float);
    int n_thread = 256;
    
    float *d_img, *d_filter, *d_inputImage;
    hipMalloc((void **)&d_img, size);
    
    hipMalloc((void**)&d_filter, sizeof(float) * filterWidth * filterWidth);
    hipMemcpy(d_filter, filter, sizeof(float) * filterWidth * filterWidth, hipMemcpyHostToDevice);
    
    hipMalloc((void**)&d_inputImage, size);
    hipMemcpy(d_inputImage, inputImage, size, hipMemcpyHostToDevice);

    // printf("\n\n-%d-%d-\n\n", (imageHeight * imageWidth + 15) / n_thread, n_thread);
    // dim3 blockSize(n_thread, n_thread);
    // dim3 numBlock(resX / n_thread, resY / n_thread);
    filterWidth /= 2;
    // printf("+%f, %f, %f, %f, %f, %f, %f, %f, %f, %f\n", filter[0], filter[1], filter[2], filter[3], filter[4], filter[5], filter[6], filter[7], filter[8], filter[9]);
    // printf("+%f, %f, %f, %f, %f, %f, %f, %f, %f, %f\n", inputImage[0], inputImage[1], inputImage[2], inputImage[3], inputImage[4], inputImage[5], inputImage[6], inputImage[7], inputImage[8], inputImage[9]);
    // printf("+%f, %f, %f, %f, %f, %f, %f, %f, %f, %f\n", outputImage[0], outputImage[1], outputImage[2], outputImage[3], outputImage[4], outputImage[5], outputImage[6], outputImage[7], outputImage[8], outputImage[9]);
    mandelKernel<<<(imageHeight * imageWidth + 255) / n_thread / 4, n_thread>>>(d_img, filterWidth, d_filter, imageHeight, imageWidth, d_inputImage);

    hipDeviceSynchronize();
    
    hipMemcpy(outputImage, d_img, size, hipMemcpyDeviceToHost);
    // printf("++%f, %f, %f, %f, %f, %f, %f, %f, %f, %f\n", inputImage[0], inputImage[1], inputImage[2], inputImage[3], inputImage[4], inputImage[5], inputImage[6], inputImage[7], inputImage[8], inputImage[9]);
    // printf("++%f, %f, %f, %f, %f, %f, %f, %f, %f, %f\n", outputImage[0], outputImage[1], outputImage[2], outputImage[3], outputImage[4], outputImage[5], outputImage[6], outputImage[7], outputImage[8], outputImage[9]);
    
    hipFree(d_img);
}
