#include <stdio.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <algorithm>
#include <time.h>

using namespace std;
#define iteration 1
#define thread_num 16
#define kernel_size 3

void cuda_err_chk(const hipError_t& e, const int& cudaError_cnt){
    if(e != hipSuccess){
        fprintf(stderr, "hipError_t in no. %d: %s\n", cudaError_cnt, hipGetErrorString(e));
        exit(EXIT_FAILURE);
    }
}

__global__ void median_filter(int* img, int* image, int h , int w){
    
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    int count = 0;
    int total = kernel_size*kernel_size;
    int p = (int)(kernel_size/2);
    int vs_x[kernel_size*kernel_size];
    int vs_y[kernel_size*kernel_size];
    int vs_z[kernel_size*kernel_size];

    if(j >= h || i >= w)return;
    
   
        count = 0;

        for(int t = 0 ; t < total ; t++){
            vs_x[t] = 0;
            vs_y[t] = 0;
            vs_z[t] = 0;
        }
        int now = ((j)*w + i)*3;
        for(int _j = -p ; _j < p+1 ; _j++){
            for (int _i = -p; _i < p+1;_i++){
                if(((j + _j) >= 0)&&((i + _i) >=0) && ((j + _j) < h) && (i + _i) < w){
                    //vs[count++] = (int)cu_img.at<cv::Vec3b>(j + _j,i + _i)[c];
                    int pixel = ((j + _j)*w +(i + _i))*3;
                    vs_x[count] = img[pixel];
                    vs_y[count] = img[pixel + 1];
                    vs_z[count++] = img[pixel + 2];
                }
            }
        }
        
        int temp;
        for(int k = 0; k < total; k++){
            for(int a = 0; a < total; a++){
                if(vs_x[k] > vs_x[a]){
                    temp = vs_x[k];
                    vs_x[k] = vs_x[a];
                    vs_x[a] = temp;
                }
                if(vs_y[k] > vs_y[a]){
                    temp = vs_y[k];
                    vs_y[k] = vs_y[a];
                    vs_y[a] = temp;
                }
                if(vs_z[k] > vs_z[a]){
                    temp = vs_z[k];
                    vs_z[k] = vs_z[a];
                    vs_z[a] = temp;
                }
            }
        }
        image[now] = vs_x[total/2];
        image[now + 1] = vs_y[total/2];

        image[now + 2] = vs_z[total/2];
             
    
}

int main(int argc, char **argv){
    
    clock_t t1, t2;
    t1 = clock();
    
    for(int i = 0; i < iteration; i++){
        int cudaError_cnt = 0;
        int *img_int;
        int *image_int;
        FILE *fp;
        fp = fopen("./4K_ultra_noise.txt", "r");
        cout<<"start"<<endl;
        int h , w, rgb;
        fscanf(fp,"%d %d %d",&h,&w,&rgb);
        img_int = (int *)malloc(h * w * 3 * sizeof(int));
        
        image_int = (int*)malloc(h * w * 3 * sizeof(int));
        cout<<h<<" "<<w<<" "<< rgb <<endl;
        for(int j = 0; j < h; j++){
            for(int k = 0; k < w; k++){
                for(int c = 0; c < 3; c++){
                        //file >> *(img_int + ((j * w) + k) * 3 + c) ;
                        fscanf(fp,"%d", (img_int + ((j * w) + k) * 3 + c) );
                        
                }
            }
            
        }
        
        t1 = clock();
        int *img_cuda;
        int *image_cuda;
        cuda_err_chk(hipMalloc((void**) &img_cuda, h * w * 3 * sizeof(int)), cudaError_cnt++);
        cuda_err_chk(hipMalloc((void**) &image_cuda, h * w * 3 * sizeof(int)), cudaError_cnt++);
        cuda_err_chk(hipMemcpy(img_cuda, img_int, h * w * 3 * sizeof(int), hipMemcpyHostToDevice), cudaError_cnt++);
        //cuda_err_chk(cudaMemcpy(image_cuda, image_int, h * w * 3 * sizeof(int), cudaMemcpyHostToDevice), cudaError_cnt++);

        
        dim3 threadsPerBlock(thread_num, thread_num);
        dim3 numBlocks((h + thread_num - 1)/thread_num, (w + thread_num - 1)/thread_num);
        median_filter<<<numBlocks, threadsPerBlock>>>(img_cuda,image_cuda, h, w);
        cuda_err_chk(hipDeviceSynchronize(), cudaError_cnt++);
        
        cuda_err_chk(hipMemcpy(image_int, image_cuda, h * w * 3 * sizeof(int), hipMemcpyDeviceToHost), cudaError_cnt++);
        t2 = clock();
        
        FILE *fp2;
        fp2 = fopen("./4k_ultra_cuda.txt", "w");
        fprintf(fp2,"%d %d %d\n" , h, w, rgb);
        for(int j = 0; j < h; j++){
            
            for(int k = 0; k < w; k++){
                for(int c = 0; c < 3; c++){
                        //file2 <<image_int[((j * w) + k) *3 + c]<<" ";
                        fprintf(fp2,"%d " , image_int[((j * w) + k) *3 + c]);
                        
                }
                fprintf(fp2,"\n");
            }
        }
        free(img_int);
        free(image_int);
        hipFree(img_cuda);
        hipFree(image_cuda);
        
        printf("%lf \n", (t2-t1)/(double)(CLOCKS_PER_SEC));
        
        fclose ( fp );
        fclose ( fp2 );
        
    }
    
    
    
}